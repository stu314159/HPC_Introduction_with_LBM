
#include <hip/hip_runtime.h>
__global__ void calcFeqD3Q19(double * fEq, const double * ux, const double * uy,
                            const double * uz, const double * rho, const int N)
{
  const double ex[19] = {0.0, 1.0, -1.0, 0.0, 0.0, 0.0, 0.0, 1.0 -1.0,
                         1.0, -1.0, 1.0, -1.0, 1.0, -1.0, 0.0, 0.0, 0.0, 0.0};
  const double ey[19] = {0.0, 0.0, 0.0, 1.0, -1.0, 0.0, 0.0, 1.0, 1.0, -1.0, 
                        -1.0, 0.0, 0.0, 0.0, 0.0, 1.0, -1.0, 1.0, -1.0};
                        
  const double ez[19] = {0.0, 0.0, 0.0, 0.0, 0.0, 1.0, -1.0, 0.0, 0.0, 0.0,
                         0.0, 1.0, 1.0, -1.0, -1.0, 1.0, 1.0, -1.0, -1.0};
                         
                         
  const double w[19] = {3.0/9.0,1.0/18.0,1.0/18.0,1.0/18.0,1.0/18.0,1.0/18.0,1.0/18.0,
                        1.0/36.0,1.0/36.0,1.0/36.0,1.0/36.0,1.0/36.0,1.0/36.0,
                        1.0/36.0,1.0/36.0,1.0/36.0,1.0/36.0,1.0/36.0,1.0/36.0};
  const int numSpd = 19;
  
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (tid < N)
  {
    double cu;
    for(int spd=0;spd<numSpd;spd++)
    {
      cu = 3.0*(ex[spd]*ux[tid]+ey[spd]*uy[tid]+ez[spd]*uz[tid]);
      fEq[N*spd+tid] = w[spd]*rho[tid]*(1.0+cu+(0.5)*cu*cu - 
                       (1.5)*(ux[tid]*ux[tid] + uy[tid]*uy[tid] 
                       + uz[tid]*uz[tid]));
    }
  }
}
